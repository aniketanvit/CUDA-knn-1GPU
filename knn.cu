/* 
* INPUT:
* m: total num of points
* m is in [10, 1000]
* n: n dimensions
* n is in [1,1000]
* k: num of nearest points
* k is in [1,10]
* V: point coordinates
* the integer elements are in [-5,5]
* OUTPUT:
* out: k nearest neighbors
*/

#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>

#define INIT_MAX 100000
void showResult(int m, int k, int *out);

extern __shared__ int SM[];

// compute the square of distance per dimension
// the kth dimension of the ith point and jth point
__device__ void computeDimDist(int i, int j, int n, int *V)
{
	int k = threadIdx.x;
	SM[k] = (V[i*n+k]-V[j*n+k])*(V[i*n+k]-V[j*n+k]);
}

// compute the square of distance of the ith point and jth point
__global__ void computeDist(int m, int n, int *V, int *D)
{
	int x = blockIdx.x;
   	int y = blockIdx.y;
	int k = threadIdx.x;
	int s;

	int i;
	int j;
	int is_valid = 0;
	int num = x*gridDim.y+y;
	int sum = 0;
	int next_sum = 0;
	int cnt;
	// identify the two points' indices 
//	__syncthreads();
//	for(cnt=0; cnt<m-1 && is_valid==0; cnt++)
	for(cnt=0; cnt<=m-1; cnt++)
	{
		sum += (m-cnt-1);
		next_sum = sum+(m-cnt-2);
//		__syncthreads();
		// check if the index of point i is valid
		if(sum<=num && num<=next_sum)
//		if(sum<=num && num<=(sum+m-cnt-2))
		{
			i = cnt;
			j = num-sum+(i+1);
			is_valid = 1;
		}
//		__syncthreads();
	}
	__syncthreads();
	// calculate the square of distance per dimensions
	// reduce duplicated calculations since d(i, j) = d(j, i)
	// also, we do not consider the trivial case of d(i, i) = 0
	// so we only compute the square distance when i < j 
	computeDimDist(i, j, n, V);
	__syncthreads();
	// use parallel reduction
	for(s=n/2; s>0; s>>=1)
	{
		if(k < s)
		{
			SM[k] += SM[k+s];
		}
		__syncthreads();
	}
	if(k == 0)
	{
		// when n is odd, the last element of SM needs to be added
		if(n > (n/2)*2)
		{
			D[i*m+j] = SM[0] + SM[n-1];
		}
		else
		{
			D[i*m+j] = SM[0];
		}
	}
}

__device__ void initSM(int m, int *D)
{
	int i = blockIdx.x;
	int j = threadIdx.x;
	if(i < j)
	{
		SM[j] = D[i*m+j];
	}
	else
	{
		SM[j] = D[j*m+i];	
	}
	__syncthreads();
}

// compute the k nearest neighbors
__global__ void knn(int m, int n, int k, int *V, int *D, int *out)
{
	int i,j;
	int temp;
	int count;
	int num;
	int dist;
	int is_duplicate;

	// find the k nearest neighbors of the point with index = blockIdx.x
	i = blockIdx.x;
	
	initSM(m, D);
	__syncthreads();

	// let the first thread select the k-min distance
	if(threadIdx.x == 0)
	{
		for(count=0; count<k; count++)
		{
			temp = INIT_MAX;
			// iterate the jth point
			for(j=0; j<m; j++)
			{
				// check whether the jth point is the same point as the ith one
				// or has already in the k-nn list
				is_duplicate = 0;
				if(j == i)
				{
					is_duplicate = 1;
				}
				for(num=0; num<count; num++)
				{
					if(out[i*k+num] == j)
					{
						is_duplicate = 1;
					}
				}
				if(!is_duplicate)
				{
					dist = SM[j];
					if(dist < temp)
					{
						temp = dist;
						out[i*k+count] = j;
					}
				}
			}
		}
	}
}

void showResult(int m, int k, int *out)
{
	int i,j;
	for(i=0; i<m; i++)
	{
		for(j=0; j<k; j++)
		{
			printf("%d ", out[i*k+j]);
			if(j == k-1)
			{
				printf("\n");
			}
			/*
			else
			{
				printf(" ");
			}
			*/
		}
	} 
} 
int main(int argc, char *argv[]) 
{ 
	int m,n,k;
	int i;
	int *V, *out;				//host copies
	int *d_V, *d_out;			//device copies
	int *D;	

	int *test_D;						

	FILE *fp_in;
	FILE *fp_out;
	if(argc != 2)
	{
		printf("Usage: knn <inputfile>\n");
		exit(1);
	}
	if((fp_in = fopen(argv[1], "r")) == NULL)
	{
		printf("Error open input file!\n");
		exit(1);
	}
	if((fp_out = fopen("time.txt", "w")) == NULL)
	{
		printf("Error open output file!\n");
		exit(1);
	}
	while(fscanf(fp_in, "%d %d %d", &m, &n, &k) != EOF)
	{
		V = (int *) malloc(m*n*sizeof(int));
		out = (int *) malloc(m*k*sizeof(int));

		test_D = (int *) malloc(m*m*sizeof(int));

		// allocate space for devices copies
		hipMalloc((void **)&d_V, m*n*sizeof(int));
		hipMalloc((void **)&d_out, m*k*sizeof(int));
		hipMalloc((void **)&D, m*m*sizeof(int));

		for(i=0; i<m*n; i++)
		{
			fscanf(fp_in, "%d", &V[i]);
		}
		// copy host values to devices copies
		hipMemcpy(d_V, V, m*n*sizeof(int), hipMemcpyHostToDevice);

		int x, y;
		// decide grid dimensions for computeDist
		if(m == (m/2)*2)
		{
			x = m/2;
			y = m-1;
		}
		else
		{
			x = (m-1)/2;
			y = m;
		}
		dim3 grid(x, y);
		// compute the execution time
		hipEvent_t start, stop;
		// create event
		hipEventCreate(&start);
		hipEventCreate(&stop);
		// record event
		hipEventRecord(start);
		// launch knn() kernel on GPU
		computeDist<<<grid, n, n*sizeof(int)>>>(m, n, d_V, D);
		hipDeviceSynchronize();
		knn<<<m, m, m*sizeof(int)>>>(m, n, k, d_V, D, d_out);
		// record event and synchronize
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		float time;
		// get event elapsed time
		hipEventElapsedTime(&time, start, stop);
		fprintf(fp_out, "GPU calculation time:%f ms\n", time);
		// copy result back to host
		hipMemcpy(out, d_out, m*k*sizeof(int), hipMemcpyDeviceToHost);

		hipMemcpy(test_D, D, m*m*sizeof(int), hipMemcpyDeviceToHost);
		showResult(m, m, test_D);
		printf("\n");

		showResult(m, k, out);
		printf("%f\n", time);
		// cleanup
		hipFree(d_V);
		hipFree(d_out);
		hipFree(D);

		free(V);
		free(out);
	}
	fclose(fp_in);
	fclose(fp_out);
	return 0;
}

