/* 
* INPUT:
* m: total num of points
* n: n dimensions
* k: num of nearest points
* V: point coordinates
* OUTPUT:
* out: k nearest neighbors
*/

#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>

#define INIT_MAX 10000000
#define TILE_WIDTH 32
#define TILE_DEPTH 128
#define MAX_BLOCK_SIZE 256
//#define MAX_BLOCK_SIZE 1024
#define MAX_PTRNUM_IN_SMEM 4096

void showResult(int m, int k, int *out);


// compute the square of distance of the ith point and jth point
__global__ void computeDist(int m, int n, int *V, int *D)
{
	__shared__ int rowVector[TILE_WIDTH][TILE_DEPTH];
	__shared__ int colVector[TILE_DEPTH][TILE_WIDTH];
	__shared__ int dist[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x;
   	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int row;
	int col;
//	int row = by*TILE_WIDTH+ty;
//	int col = bx*TILE_WIDTH+tx;
	int px;
	int py;	

	for(py=ty; py<TILE_WIDTH; py+=blockDim.y)
	{
		for(px=tx; px<TILE_WIDTH; px+=blockDim.x)
		{
		
			row = by*TILE_WIDTH+py;
			col = bx*TILE_WIDTH+px;
			dist[py][px] = 0;
			__syncthreads();
		
			for(int i=0; i<(int)(ceil((float)n/TILE_DEPTH)); i++)
			{
				for(int j=0; j<TILE_DEPTH; j++)
				{
					rowVector[py][j] = V[row*n+i*TILE_DEPTH+j];
				}
				for(int j=0; j<TILE_DEPTH; j++)
				{		
					colVector[j][px] = V[col*n+i*TILE_DEPTH+j];
				}
				__syncthreads();
		
				for(int j=0; j<TILE_DEPTH; j++)
				{
					dist[py][px] += (rowVector[py][j]-colVector[j][px])*(rowVector[py][j]-colVector[j][px]);
				}
				__syncthreads();
			}
			D[row*m+col] = dist[py][px];
		}
	}
}

extern __shared__ int SMem[];

__device__ void initSMem(int m, int k, int count, int *D, int *out)
{
	int i = blockIdx.x;
	int j = threadIdx.x;
	if(j == i)
	{
		SMem[i] = INIT_MAX;
	}
	else
	{
		SMem[j] = D[i*m+j];
	}
	if(j < count)
	{
		SMem[ out[i*k+j] ] = INIT_MAX;
	}
	//index
	SMem[j+m] = j;
}

__device__ int findMin(int m, int k, int count, int *D, int *out)
{
	int i = blockIdx.x;
  	int tid = threadIdx.x;

	int s = blockDim.x/2;
	int resultValue = INIT_MAX;
	int resultIndex = INIT_MAX;
	int indexBase = (m<MAX_PTRNUM_IN_SMEM)? m: MAX_PTRNUM_IN_SMEM;
	
	for(int num=0; num<m; num+=MAX_PTRNUM_IN_SMEM)
	{
//		for(int j=tid; j<indexBase; j+=blockDim.x)
		int range = indexBase/blockDim.x;
		int start = tid*range;
		int end = (tid+1)*range;
		for(int j=start; j<end; j++)
		{
			if(j+num == i)
			{
				SMem[j] = INIT_MAX;
			}
			else
			{
				SMem[j] = D[i*m+num+j];
			}
			//index
			SMem[indexBase+j] = j+num;
			__syncthreads();
/*
			if(tid < count)
			{
				if(out[i*k+tid]-num>0 && out[i*k+tid]-num < indexBase)
				{
					SMem[ out[i*k+tid]-num ] = INIT_MAX;
				}
			}
			__syncthreads();
*/
		}
//		__syncthreads();
		for(int j=0; j<count; j++)
		{
			if(out[i*k+j]-num>=0 && out[i*k+j]-num<indexBase)
			{
				SMem[ out[i*k+j]-num ] = INIT_MAX;
			}
			__syncthreads();
		}
		__syncthreads();
/*
		if(tid < count)
		{
			if(out[i*k+tid]-num>0 && out[i*k+tid]-num < indexBase)
			{
				SMem[ out[i*k+tid]-num ] = INIT_MAX;
			}
//			SMem[ out[i*k+tid] ] = INIT_MAX;
		}
		__syncthreads();
*/
		for(s=indexBase/2; s>0; s>>=1) 
		{
			for(int j=tid; j<indexBase; j+=blockDim.x)
			{
				if(j < s) 
				{
					if(SMem[j] == SMem[j+s])
					{
						if(SMem[indexBase+j] > SMem[indexBase+j+s])
						{
							SMem[indexBase+j] = SMem[indexBase+j+s];
						}
					}
					else if(SMem[j] > SMem[j+s])
					{
						SMem[j] = SMem[j+s];
						SMem[indexBase+j] = SMem[indexBase+j+s];
					}
				}
				__syncthreads();
			}
		}
	
		if(resultValue == SMem[0])
		{
			if(resultIndex > SMem[indexBase])
			{
				resultIndex = SMem[indexBase];
			}
		} 
		else if (resultValue > SMem[0])
		{
			resultValue = SMem[0];
			resultIndex = SMem[indexBase];
		}
		__syncthreads();
	}
	return resultIndex;

}

// compute the k nearest neighbors
__global__ void knn(int m, int k, int *V, int *D, int *out)
{
	int i;
	int count;

	i = blockIdx.x;
	__syncthreads();
	for(count=0; count<k; count++)
	{
		out[i*k+count] = findMin(m, k, count, D, out);
		__syncthreads();
	}
}

void showD(int m, int *D)
{
	int i,j;
	printf("D:\n");
	for(i=0; i<m; i++)
	{
		for(j=0; j<m; j++)
		{
			printf("%d ", D[i*m+j]);
			if(j == m-1)
			{
				printf("\n");
			}	
		}    	
	}        	
	printf("D:\n");
}            	

void showResult(int m, int k, int *out)
{
	int i,j;
	for(i=0; i<m; i++)
	{
		for(j=0; j<k; j++)
		{
			printf("%d ", out[i*k+j]);
			if(j == k-1)
			{
				printf("\n");
			}	
		}    	
	}        	
}            	
int main(int argc, char *argv[]) 
{ 
	int m,n,k;
	int i;
	int *V, *out;				//host copies
	int *d_V, *d_out;			//device copies
	int *D;						

//int *h_D;

	FILE *fp;
	if(argc != 2)
	{
		printf("Usage: knn <inputfile>\n");
		exit(1);
	}
	if((fp = fopen(argv[1], "r")) == NULL)
	{
		printf("Error open input file!\n");
		exit(1);
	}
	while(fscanf(fp, "%d %d %d", &m, &n, &k) != EOF)
	{
		V = (int *) malloc(m*n*sizeof(int));
		out = (int *) malloc(m*k*sizeof(int));
		for(i=0; i<m*n; i++)
		{
			fscanf(fp, "%d", &V[i]);
		}

//h_D = (int *) malloc(m*m*sizeof(int));
//cudaDeviceSetCacheConfig(cudaFuncCachePreferShared);	

		// compute the execution time
		hipEvent_t start, stop;
		// create event
		hipEventCreate(&start);
		hipEventCreate(&stop);
		// record event
		hipEventRecord(start);

		// allocate space for devices copies
		hipMalloc((void **)&d_V, m*n*sizeof(int));
		hipMalloc((void **)&d_out, m*k*sizeof(int));
		hipMalloc((void **)&D, m*m*sizeof(int));

		// copy host values to devices copies
		hipMemcpy(d_V, V, m*n*sizeof(int), hipMemcpyHostToDevice);

		int gridDimX = (int)(ceil((float)m/TILE_WIDTH));
		int gridDimY = (int)(ceil((float)m/TILE_WIDTH));

		dim3 grid(gridDimX, gridDimY);
//		dim3 grid(m, m);

		dim3 block(TILE_WIDTH/2, TILE_WIDTH/2);
//		dim3 block(TILE_WIDTH, TILE_WIDTH);

		// launch knn() kernel on GPU
		computeDist<<<grid, block>>>(m, n, d_V, D);
//		computeDist<<<grid, n, n*sizeof(int)>>>(m, n, d_V, D);
		hipDeviceSynchronize();

//cudaMemcpy(h_D, D, m*m*sizeof(int), cudaMemcpyDeviceToHost);
//showD(m, h_D);

		int threadNum = (m<MAX_BLOCK_SIZE)? m: MAX_BLOCK_SIZE;
//		knn<<<m, m, 2*m*sizeof(int)>>>(m, k, d_V, D, d_out);

		int ptrNumInSMEM = (m<MAX_PTRNUM_IN_SMEM)? m: MAX_PTRNUM_IN_SMEM;
		knn<<<m, threadNum, 2*ptrNumInSMEM*sizeof(int)>>>(m, k, d_V, D, d_out);
//		knn<<<m, threadNum, 2*m*sizeof(int)>>>(m, k, d_V, D, d_out);

		// copy result back to host
		hipMemcpy(out, d_out, m*k*sizeof(int), hipMemcpyDeviceToHost);

		// cleanup
		hipFree(d_V);
		hipFree(d_out);
		hipFree(D);

		// record event and synchronize
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		float time;
		// get event elapsed time
		hipEventElapsedTime(&time, start, stop);

		showResult(m, k, out);
		if(m == 1024) {
			printf("SMALL:");
		} else if(m == 4096) {
			printf("MIDDLE:");
		} else if(m == 16384) {
			printf("LARGE:");
		}
		printf("%f\n", time);

		free(V);
		free(out);
//free(h_D);
	}
	fclose(fp);
	return 0;
}

